#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>

#include "common.h"
#include "flow1.h"
#include "flow3.h"

int main(int argc, char **argv)
{
	if (argc != 4)
	{
		printf("Usage: %s 1 2 logfile.pre\n", argv[0]);
		printf("1: number of runs\n");
		printf("2: number of threads per block (for GPU version)\n");
		return 0;
	}
	
	n_runs = atoi(argv[1]);
	n_threads_per_block = atoi(argv[2]);
	
	read_log(argv[argc-1]);
	
	n_matrix = n_tasks*n_tasks;

	int* matrix1 = flow1();
//	print_matrix(matrix1, n_matrix);

	int* matrix3 = flow3();
//	print_matrix(matrix3, n_matrix);

	printf("Check matrix: ");
	if (memcmp(matrix1, matrix3, n_matrix*sizeof(int)) == 0)
	{
		printf("OK");
	}
	else
	{
		printf("Not OK");
	}
	printf("\n");

	free(matrix1);
	free(matrix3);

	free_log();

	return 0;
}
